#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
*/
#include <iostream>
#include <tuple>

#include<stdio.h>

#include <cuspatial_test/base_fixture.hpp>
#include <cuspatial_test/vector_factories.cuh>
#include <cuspatial_test/vector_equality.hpp>

#include <cuspatial/geometry/segment.cuh>
#include <cuspatial/geometry/vec_2d.hpp>

#include  <cuspatial/area.cuh>
#include <cuspatial/error.hpp>


#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_scalar.hpp>
#include <rmm/device_vector.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>
#include <rmm/mr/device/device_memory_resource.hpp>


#include <gtest/gtest.h>

#include <initializer_list>


#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/iterator/counting_iterator.h>

// template <typename MultiPointRange>
// __global__ void print_kernel(MultiPointRange point_rng, std::size_t n){
//   if(threadIdx.x == 0){
//     for (auto ptr = point_rng.point_begin(); ptr < point_rng.point_end(); ptr++) {
//         printf("%lf", ptr[0].x); 
//     }
//   }
// }

using namespace cuspatial;
using namespace cuspatial::test;

template <typename MultiPointRange>
struct compute_area_functor {
    using T = typename MultiPointRange::element_t;

    MultiPointRange multipoints;
    
    compute_area_functor(MultiPointRange multipoints) : multipoints(multipoints)
    {
    }

    template <typename IndexType>
    __device__ T operator()(IndexType pidx) {
      T area = 0.0;

      if(pidx < multipoints.num_points() - 1){
        vec_2d<T> point_1 = multipoints.point(pidx);
        vec_2d<T> point_2 = multipoints.point(pidx+1);
        
        area = (point_1.x*point_1.y - point_1.y*point_2.x)/2;

      }
      
      return area;
    }
};


template <typename T>
struct AreaTest : public BaseFixture {
  void run_multipolygon_area(std::initializer_list<std::size_t> multipolygon_geometry_offsets,
           std::initializer_list<std::size_t> multipolygon_part_offsets,
           std::initializer_list<std::size_t> multipolygon_ring_offsets,
           std::initializer_list<vec_2d<T>> multipolygon_coordinates,
           std::initializer_list<T> expected)
  {
      using Location = vec_2d<T>;
      auto multipolygon = make_multipolygon_array(multipolygon_geometry_offsets,
                                                  multipolygon_part_offsets,
                                                  multipolygon_ring_offsets,
                                                  multipolygon_coordinates);
      
      auto rng = multipolygon.range().as_multipoint_range();
      unsigned long int num_points = rng.num_points();
      auto out = rmm::device_vector<T>{num_points};

      compute_area_functor functor(rng);

      thrust::counting_iterator<int> iter(0);

      thrust::transform(iter, iter+rng.num_points(), out.begin(), functor);
      
      thrust::host_vector<T> h_out(out);

      for(auto it = h_out.begin(); it != h_out.end(); ++it){
        std::cout << "Element " << *it << std::endl;
      }
      // auto test = multipolygon.to_host();
      // auto [geometry_offsets, part_offsets, ring_offsets, coordinates] = multipolygon.to_host();

      // for (size_t i = 0; i < coordinates.size(); i++){
      //   std::cout << "Element at index " << i << ": " << coordinates[i] << std::endl;
      // }
      // create device vector
      // auto area = rmm::device_vector<T>({0, 1});

      // auto d_expected = make_device_uvector(expected, stream(), mr());

      // cuspatial::area(rng, area, rmm::cuda_stream_default);
      
  }
};

// float and double are logically the same but would require separate tests due to precision.
using TestTypes = ::testing::Types<float, double>;

TYPED_TEST_CASE(AreaTest, TestTypes);


TYPED_TEST(AreaTest, LinestringAreaTest)
{
  // linestring has no area
  // CUSPATIAL_RUN_TEST(this->run_multipolygon_area, 
  //                    {}, //
  //                    {},
  //                    {},
  //                    {{}},
  //                    {});   // expected areas

}


TYPED_TEST(AreaTest, PolygonSquareAreaTest)
{
  CUSPATIAL_RUN_TEST(this->run_multipolygon_area, 
                     {0, 1},    // geometry offsets
                     {0, 1},    // part offsets
                     {0, 4},    // ring offsets
                     {{0, 0}, {1, 0}, {1, 1}, {0, 0}},  // coordinates
                     {1}); // expected areas

}

TYPED_TEST(AreaTest, MultiPolygonHexagonAreaTest)
{
  CUSPATIAL_RUN_TEST(this->run_multipolygon_area, 
                     {0, 1},    // geometry offsets
                     {0, 1},    // part offsets
                     {0, 6},    // ring offsets
                     {{0, 0}, {1, 0}, {1, 1}, {-1, -2}, {0, -1}, {0, 0}},  // coordinates
                     {1}); // expected areas

}


